#include "hip/hip_runtime.h"
#include "sim.h"
#include <cstdlib>
#include "interop.h"
#include "kernel.h"

Simulation::Simulation(int width, int height, int max_points)
: width(width), height(height), max_point_count(max_points), point_count(0)
{
    createGLTextureForCUDA(&gl_tex, &cuda_tex_resource, width, height);

    hipMalloc((void**)&cuda_dev_render_buffer, width * height * sizeof(float));

	hipMalloc((void**)&d_points, max_points * sizeof(float4) * 2);
	hipMalloc((void**)&d_transform, sizeof(float4) * 4);

	amp = 1.0f;
}

Simulation::~Simulation()
{
	hipFree(this->d_points);
	hipFree(this->d_pattern);
	hipFree(this->d_transform);
}

void Simulation::generateImage(void)
{
	launch_transform(64, 64, 
		(float4*)d_points, 
		(float4*)d_points + max_point_count,
		(float4*)d_transform, 
		amp,
		point_count);

    launch_kernel(N_BLOCKS, N_THREADS,
		(float*)cuda_dev_render_buffer,
		(float4*)d_points + max_point_count,
		point_count);
	
	hipArray * texture_ptr;
	CUDA_CALL(hipGraphicsMapResources(1, &cuda_tex_resource, 0));
	CUDA_CALL(hipGraphicsSubResourceGetMappedArray(&texture_ptr, cuda_tex_resource, 0, 0));

	int size_tex_data = width * height * sizeof(float);

	CUDA_CALL(hipMemcpyToArray(
		texture_ptr, 0, 0, 
		cuda_dev_render_buffer, 
		size_tex_data, 
		hipMemcpyDeviceToDevice));
	CUDA_CALL(hipGraphicsUnmapResources(1, &cuda_tex_resource, 0));
}

extern double GLFW_TIME;

void Simulation::setPoints(void * points, int count)
{
	if(count > this->max_point_count)
		count = this->max_point_count;

	this->point_count = count;

	hipMemcpy(d_points, points, count * sizeof(float4), hipMemcpyHostToDevice);
}

void Simulation::SetTransformation(Ruined::Math::Matrix mat)
{
	hipMemcpy(d_transform, mat.m, sizeof(float4) * 4, hipMemcpyHostToDevice);
}