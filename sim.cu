#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "pattern.h"

#define POINT_COUNT     3

#define CUDART_2PI      6.2831853071795865f
#define HIP_PI       3.1415926535897931f
#define CUDART_PIO6     0.52359877559829887f

#define ARRAY_WIDTH     (684)
#define ARRAY_HEIGHT    (608)
#define ARRAY_SIZE      (ARRAY_WIDTH * ARRAY_HEIGHT)

#define ARRAY_WIDTH_INV     (1.0f / ARRAY_WIDTH)  //0.0014619883040935673
#define ARRAY_HEIGHT_INV    (1.0f / ARRAY_HEIGHT) //0.0016447368421052632

#define PATTERN_WIDTH   6571.8e-6f
#define PATTERN_HEIGHT  3699e-6f
#define N_BLOCKS    256     //
#define N_THREADS   512     // Threads per block

#define LAMBDA          632.8e-9f
#define LAMBDA_INV      (1.0f / LAMBDA) //1.580278128950695e6
#define TWO_LAMBDA_INV  (2.0f / LAMBDA) //3.160556257901391e6
#define REF_BEAM_ANGLE  (0.00565003f)    // ~0.33 degrees

#define PLANE_CONST (TWO_LAMBDA_INV * sin(REF_BEAM_ANGLE))
#define VAL_CONST   (TWO_LAMBDA_INV)

__device__ float plane(float x)
{
    return cospif(PLANE_CONST * x);
}

__device__ float distance(float2 uv, float3 point)
{
    return norm3df(uv.x - point.x, uv.y - point.y, point.z);
}

__device__ float val(float2 uv, float3 point)
{
    float d = distance(uv, point);
    return sinpif(remainderf(d, LAMBDA) * VAL_CONST);
}

__device__ float intensity(float2 uv, float3 point)
{
    float x = uv.x - point.x;
    float y = uv.y - point.y;
    return 1.0f / (x * x + y * y + point.z * point.z);
}

__global__ void simulation(float * pattern, float4 * points, int count)
{
    for(int x = threadIdx.x; x < ARRAY_WIDTH; x += blockDim.x)
    {
        for(int y = blockIdx.x; y < ARRAY_HEIGHT; y+= gridDim.x)
        {
            float2 uv = make_float2(x * ARRAY_WIDTH_INV * PATTERN_WIDTH, y * ARRAY_HEIGHT_INV * PATTERN_HEIGHT);
            int index = y * ARRAY_WIDTH + x;

            pattern[index]= plane(uv.x);

            for(int i = 0; i < count; i++)
            {
                float3 point = make_float3(points[i].x, points[i].y, points[i].z);
                pattern[index] += points[i].w * intensity(uv, point) * val(uv, point);
            }
        }
    }

    return;
}

int main(void)
{
    float * d_pattern;

    float4 * h_points;
    float4 * d_points;

    dim3 dimGrid(N_BLOCKS);
    dim3 dimBlock(N_THREADS);

    pattern pat(ARRAY_WIDTH, ARRAY_HEIGHT);

    h_points  =  (float4*)malloc(POINT_COUNT * sizeof(float4));

    hipMalloc((void**)&d_pattern,  ARRAY_SIZE * sizeof(float));
    hipMalloc((void**)&d_points,   POINT_COUNT* sizeof(float4));

    h_points[0] = make_float4(PATTERN_WIDTH * 0.50f, PATTERN_HEIGHT * 0.25f, 0.30f, 0.33f);
    h_points[1] = make_float4(PATTERN_WIDTH * 0.50f, PATTERN_HEIGHT * 0.50f, 0.30f, 0.33f);
    h_points[2] = make_float4(PATTERN_WIDTH * 0.50f, PATTERN_HEIGHT * 0.75f, 0.30f, 0.33f);

    hipMemcpy(d_points, h_points, POINT_COUNT * sizeof(float4), hipMemcpyHostToDevice);

    // call method with dumb syntax
    simulation<<<dimGrid, dimBlock>>>(d_pattern, d_points, POINT_COUNT);

    hipDeviceSynchronize();

    hipMemcpy(pat.data, d_pattern, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    free(h_points);
    hipFree(d_pattern);
    hipFree(d_points);

    pat.export_bmp("sim_f32.bmp");
    pat.save("sim_f32.out");

    return 0;
}
